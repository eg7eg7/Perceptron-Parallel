#include "hip/hip_runtime.h"

#include <stdio.h>
#include <omp.h>
#include <math.h>
#include "cudaKernel.h"
#include "Perceptron.h"

hipError_t memcpyDoubleArrayToHost(double **dest, double **src, int n) {
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMemcpy(*dest, *src, n * sizeof(double), hipMemcpyDeviceToHost);
	CHECK_ERRORS(cudaStatus, "hipMemcpy - double failed\n", hipErrorUnknown);
	return cudaStatus;
}

hipError_t memcpy_double_array_to_device(double **dest, double **src, int n) {
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMemcpy(*dest, *src, n * sizeof(double), hipMemcpyHostToDevice);
	CHECK_ERRORS(cudaStatus, "hipMemcpy - double failed\n", hipErrorUnknown);
	return cudaStatus;
}

hipError_t memcpy_point_array_to_device(Point **dest, Point **src, int n) {
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMemcpy(*dest, *src, n * sizeof(Point), hipMemcpyHostToDevice);
	CHECK_ERRORS(cudaStatus, "hipMemcpy - Point failed\n", hipErrorUnknown);
	return cudaStatus;
}

__global__ void count_correct_points_kernel(int *result, int *sum_results, int size) {
	int i, index = threadIdx.x;
	sum_results[index] = 0;
	int chunk_size = NUM_CUDA_CORES;
	int start_index = threadIdx.x * chunk_size;

	for (i = start_index; i < start_index + chunk_size; i++) {
		if (i >= size)
			break;
		if (result[i] != POINT_CORRECT) {
			sum_results[index]++;
		}
	}
}

__device__ void mult_scalar_with_vector_device(double* vector, int dim, double scalar, double* result_vector) {
	for (int i = 0; i < dim; i++)
		result_vector[i] = vector[i] * scalar;
}

__device__ void add_vector_to_vector_device(double* vector1, double* vector2, int dim, double* result_vector) {
	for (int i = 0; i < dim; i++)
		result_vector[i] = vector1[i] + vector2[i];
}
__device__ int sign_device(double val)
{
	if (val >= 0)
		return SET_A;
	return SET_B;
}

__global__ void sum_count_results_kernel(int *sum_results, int size) {
	int sum = 0;
	for (int i = 0; i < size; i++)
	{
		sum += sum_results[i];
	}
	sum_results[0] = sum;
}


__device__ double mult_vector_with_vector_device(double* vector1, double* vector2, int dim) {
	double result = vector1[0] * vector2[0];
	for (int i = 1; i < dim; i++)
		result += vector1[i] * vector2[i];
	return result;
}

__global__ void f_on_GPU_kernel(int *result, Point* points, double* W, int N, int K) {
	int index = threadIdx.x + blockIdx.x * NUM_CUDA_CORES;
	if (index >= N)
		return;
	double val = mult_vector_with_vector_device(points[index].x, W, K + 1);
	if (sign_device(val) != points[index].set)
		result[index] = index;
	else
		result[index] = POINT_CORRECT;

}
hipError_t set_device()
{
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipSetDevice(0);
	CHECK_ERRORS(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?", hipErrorUnknown);
	return cudaStatus;
}
hipError_t cuda_malloc_double_by_size(double** arr, int arr_size)
{
	set_device();
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMalloc((void**)arr, arr_size * sizeof(double));
	CHECK_ERRORS(cudaStatus, "hipMalloc failed!", hipErrorUnknown);
	return cudaStatus;
}
hipError_t cuda_malloc_point_by_size(Point** arr, int arr_size)
{
	set_device();
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMalloc((void**)arr, arr_size * sizeof(Point));
	CHECK_ERRORS(cudaStatus, "hipMalloc failed!", hipErrorUnknown);
	return cudaStatus;
}

hipError_t free_cuda_point_array(Point** dev_points) {
	hipError_t cudaStatus = hipSuccess;
	set_device();
	Point point_zero;
	cudaStatus = hipMemcpy(&point_zero, (*dev_points), sizeof(Point), hipMemcpyDeviceToHost);
	CHECK_ERRORS(cudaStatus, "cudaMemCpy failed!", hipErrorUnknown);

	//freeing dev_points[0].x will free the rest of the points memory as well
	cudaStatus = hipFree(point_zero.x);
	CHECK_ERRORS(cudaStatus, "hipFree failed!", hipErrorUnknown);
	cudaStatus = hipFree(*dev_points);
	CHECK_ERRORS(cudaStatus, "hipFree failed!", hipErrorUnknown);
	return cudaStatus;
}
hipError_t cuda_malloc_and_free_pointers_from_quality_function(int N, int K, int num_blocks, double** W_dev, int** device_results, int** sum_results, int malloc_flag)
{
	static int is_last_malloc_flag = FREE_MALLOC_FLAG;
	static double *W_dev_p = 0;
	static int *device_results_p = 0, *sum_results_p = 0;
	hipError_t cudaStatus = hipSuccess;

	set_device();
	if (!is_last_malloc_flag && malloc_flag == MALLOC_FLAG)
	{
		cudaStatus = hipMalloc((void**)W_dev, sizeof(double)*(K + 1));
		CHECK_ERRORS(cudaStatus, "hipMalloc failed!\n", hipErrorUnknown);

		cudaStatus = hipMalloc((void**)device_results, sizeof(int)*N);
		CHECK_ERRORS(cudaStatus, "hipMalloc failed!\n", hipErrorUnknown);
		cudaStatus = hipMalloc((void**)sum_results, sizeof(int)*num_blocks);
		CHECK_ERRORS(cudaStatus, "hipMalloc failed!\n", hipErrorUnknown);

		W_dev_p = *W_dev;
		device_results_p = *device_results;
		sum_results_p = *sum_results;
		is_last_malloc_flag = MALLOC_FLAG;
	}
	else if (is_last_malloc_flag && malloc_flag == FREE_MALLOC_FLAG)
	{
		cudaStatus = hipFree(W_dev_p);
		CHECK_ERRORS(cudaStatus, "hipFree failed!\n", hipErrorUnknown);
		cudaStatus = hipFree(device_results_p);
		CHECK_ERRORS(cudaStatus, "hipFree failed!\n", hipErrorUnknown);
		cudaStatus = hipFree(sum_results_p);
		CHECK_ERRORS(cudaStatus, "hipFree failed!\n", hipErrorUnknown);
		is_last_malloc_flag = FREE_MALLOC_FLAG;
	}
	return cudaStatus;
}


hipError_t get_quality_with_GPU(Point* points, double* W, int N, int K, double* q) {
	static int *device_results, *sum_results;
	static double *W_dev;

	int count;
	int num_blocks = (int)ceil(N / (double)NUM_CUDA_CORES);
	hipError_t cudaStatus = hipSuccess;

	cuda_malloc_and_free_pointers_from_quality_function(N, K, num_blocks, &W_dev, &device_results, &sum_results, MALLOC_FLAG);

	memcpy_double_array_to_device(&W_dev, &W, K + 1);

	/*Do f on all points with adjusted W*/
	f_on_GPU_kernel <<<num_blocks, NUM_CUDA_CORES >>> (device_results, points, W_dev, N, K);
	CHECK_AND_SYNC_ERRORS("fOnGPUKernel launch failed\n");

	/*count number of correct points in each block*/
	count_correct_points_kernel <<<1, num_blocks >>> (device_results, sum_results, N);
	CHECK_AND_SYNC_ERRORS("sumResultsKernel launch failed\n");

	/*count of incorrect points in sum_results[0] - sum of sums from previous function*/
	sum_count_results_kernel <<<1, 1 >>> (sum_results, num_blocks);
	CHECK_AND_SYNC_ERRORS("adjustW_with_faulty_point launch failed\n");

	cudaStatus = hipMemcpy(&count, &(sum_results[0]), sizeof(int), hipMemcpyDeviceToHost);
	CHECK_ERRORS(cudaStatus, "Cudamemcpy failed\n", hipErrorUnknown);

	*q = (count / (double)N);
	return cudaStatus;
}