#include "hip/hip_runtime.h"

#include <stdio.h>
#include <omp.h>
#include <math.h>
#include "cudaKernel.h"
#include "Perceptron.h"

hipError_t memcpyDoubleArrayToHost(double **dest, double **src, int n) {
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMemcpy(*dest, *src, n * sizeof(double), hipMemcpyDeviceToHost);
	CHECK_ERRORS(cudaStatus, "hipMemcpy - double failed\n", hipErrorUnknown)
		return cudaStatus;
}

hipError_t memcpyDoubleArrayToDevice(double **dest, double **src, int n) {
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMemcpy(*dest, *src, n * sizeof(double), hipMemcpyHostToDevice);
	CHECK_ERRORS(cudaStatus, "hipMemcpy - double failed\n", hipErrorUnknown)
	return cudaStatus;
}

hipError_t memcpyPointArrayToDevice(Point **dest, Point **src, int n) {
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMemcpy(*dest, *src, n * sizeof(Point), hipMemcpyHostToDevice);
	
	CHECK_ERRORS(cudaStatus, "hipMemcpy - Point failed\n", hipErrorUnknown)
	return cudaStatus;
}


__global__ void countCorrectPointsKernel(int *result, int *sum_results, int size) {
	int i, index = threadIdx.x;
	sum_results[index] = 0;
	int chunk_size = NUM_CUDA_CORES;
	int start_index = threadIdx.x * chunk_size;

	for (i = start_index; i < start_index + chunk_size; i++) {
		if (i >= size)
			break;
		if (result[i] != POINT_CORRECT) {
			sum_results[index]++;
		}
	}

}

__global__ void findFirstIncorrectPointInBlockKernel(int *result, int *sum_results, int size) {
	int i, index = threadIdx.x;
	sum_results[index] = POINT_CORRECT;
	int chunk_size = NUM_CUDA_CORES;
	int start_index = threadIdx.x * chunk_size;

	for (i = start_index; i < start_index + chunk_size; i++) {
		if (i >= size)
			break;
		if (result[i] != POINT_CORRECT) {
			sum_results[index] = result[i];
			break;
		}
	}

}

__device__ void mult_scalar_with_vector_device(double* vector, int dim, double scalar, double* result_vector) {
	for (int i = 0; i < dim; i++)
		result_vector[i] = vector[i] * scalar;
}

__device__ void add_vector_to_vector_device(double* vector1, double* vector2, int dim, double* result_vector) {
	for (int i = 0; i < dim; i++)
		result_vector[i] = vector1[i] + vector2[i];
}
__device__ int sign_device(double val)
{
	if (val >= 0)
		return SET_A;
	return SET_B;
}
__device__ void device_adjustW(double* W, double* temp_vector, Point* point, int K, double alpha) {
	double val = mult_vector_with_vector_device((*point).x, W, K + 1);
	int sign = sign_device(val);
	mult_scalar_with_vector_device((*point).x, K + 1, alpha*(-sign), temp_vector);
	add_vector_to_vector_device(W, temp_vector, K + 1, W);

}
__global__ void sumCountResultsKernel(int *sum_results, int size) {
	int sum=0;
	for (int i = 0; i < size; i++)
	{
		sum += sum_results[i];
	}
	sum_results[0] = sum;
}

__global__ void adjustW_with_faulty_point(int *faulty_points,int size,Point* points, double* W,double* temp_vector,int K,double alpha) {

	int index;
	for (int i = 0; i < size; i++)
	{
		index = faulty_points[i];
		if (index != POINT_CORRECT)
		{
			//adjust W and return 
			device_adjustW(W,temp_vector, &(points[index]),K,alpha);
			faulty_points[0] = W_ADJUSTED;
			return;
		}
	}
	faulty_points[0] = ALL_POINTS_CORRECT;
}

__device__ double mult_vector_with_vector_device(double* vector1, double* vector2, int dim) {
	double result = vector1[0] * vector2[0];
	for (int i = 1; i < dim; i++)
		result += vector1[i] * vector2[i];
	return result;
}

__global__ void fOnGPUKernel(int *result, Point* points,double* W, int N,int K) {

	int index = threadIdx.x + blockIdx.x * NUM_CUDA_CORES;
	
	if (index >= N)
		return;
	double val = mult_vector_with_vector_device(points[index].x, W, K+1);
	if (sign_device(val) != points[index].set)
		result[index] = index;
	else
		result[index] = POINT_CORRECT;

}
hipError_t setDevice()
{
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipSetDevice(0);
	CHECK_ERRORS(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?", hipErrorUnknown)
	return cudaStatus;
}
hipError_t cudaMallocDoubleBySize(double** arr, int arr_size)
{
	setDevice();
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMalloc((void**)arr, arr_size * sizeof(double));
	CHECK_ERRORS(cudaStatus, "hipMalloc failed!", hipErrorUnknown)
	return cudaStatus;
}
hipError_t cudaMallocPointBySize(Point** arr, int arr_size)
{
	setDevice();
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipMalloc((void**)arr, arr_size * sizeof(Point));
	CHECK_ERRORS(cudaStatus, "hipMalloc failed!", hipErrorUnknown)
	return cudaStatus;
}

hipError_t freeCudaPointArray(Point** dev_points) {
	hipError_t cudaStatus = hipSuccess;
	setDevice();
	Point point0;
	cudaStatus = hipMemcpy(&point0, (*dev_points), sizeof(Point), hipMemcpyDeviceToHost);
	CHECK_ERRORS(cudaStatus, "cudaMemCpy failed!", hipErrorUnknown)

	//freeing dev_points[0].x will free the rest of the points memory as well
	cudaStatus = hipFree(point0.x);
	CHECK_ERRORS(cudaStatus, "hipFree failed!", hipErrorUnknown)
	cudaStatus = hipFree(*dev_points);
	CHECK_ERRORS(cudaStatus, "hipFree failed!", hipErrorUnknown)
	return cudaStatus;
}
hipError_t cudaMallocAndFreePointersFromQualityFunction(int N, int K, int num_blocks, double** W_dev, double** W_dev_temp,int** device_results, int** sum_results, int malloc_flag)
{
	static int isLastMalloc = FREE_MALLOC_FLAG;
	static double *W_dev_p = 0, *W_dev_temp_p = 0;
	static int *device_results_p=0,*sum_results_p=0;
	hipError_t cudaStatus = hipSuccess;
	
	setDevice();
	if (!isLastMalloc && malloc_flag==MALLOC_FLAG)
	{
	cudaStatus = hipMalloc((void**)W_dev, sizeof(double)*(K + 1));
	CHECK_ERRORS(cudaStatus, "hipMalloc failed!\n", hipErrorUnknown)
	cudaStatus = hipMalloc((void**)W_dev_temp, sizeof(double)*(K + 1));
	CHECK_ERRORS(cudaStatus, "hipMalloc failed!\n", hipErrorUnknown)
	cudaStatus = hipMalloc((void**)device_results, sizeof(int)*N);
	CHECK_ERRORS(cudaStatus, "hipMalloc failed!\n", hipErrorUnknown)
	cudaStatus = hipMalloc((void**)sum_results, sizeof(int)*num_blocks);
	CHECK_ERRORS(cudaStatus, "hipMalloc failed!\n", hipErrorUnknown)

	W_dev_p = *W_dev;
	W_dev_temp_p = *W_dev_temp;
	device_results_p = *device_results;
	sum_results_p = *sum_results;
	isLastMalloc = MALLOC_FLAG;
	}
	else if(isLastMalloc && malloc_flag==FREE_MALLOC_FLAG)
	{
		cudaStatus = hipFree(W_dev_p);
		CHECK_ERRORS(cudaStatus, "hipFree failed!\n", hipErrorUnknown)
		cudaStatus = hipFree(W_dev_temp_p);
		CHECK_ERRORS(cudaStatus, "hipFree failed!\n", hipErrorUnknown)
		cudaStatus = hipFree(device_results_p);
		CHECK_ERRORS(cudaStatus, "hipFree failed!\n", hipErrorUnknown)
		cudaStatus = hipFree(sum_results_p);
		CHECK_ERRORS(cudaStatus, "hipFree failed!\n", hipErrorUnknown)
		isLastMalloc = FREE_MALLOC_FLAG;
	}
	return cudaStatus;
}
hipError_t syncAndCheckErrors(const char* msg)
{
	hipError_t cudaStatus = hipGetLastError();
	CHECK_ERRORS(cudaStatus, msg, hipErrorUnknown)
	cudaStatus = hipDeviceSynchronize();
	CHECK_ERRORS(cudaStatus, "Cuda sync failed\n", hipErrorUnknown)
	return cudaStatus;
}
hipError_t get_quality_with_alpha_GPU(Point* points, double alpha, double* W, int N, int K, int LIMIT, double* q) {
	static int *device_results,*sum_results;
	static double *W_dev, *W_dev_temp;

	int flag_sum_results = W_ADJUSTED;
	int num_blocks = (int)ceil(N / (double)NUM_CUDA_CORES);
	double t1, t2;
	hipError_t cudaStatus = hipSuccess;
	
	cudaMallocAndFreePointersFromQualityFunction(N,K,num_blocks,&W_dev,&W_dev_temp,&device_results,&sum_results,MALLOC_FLAG);

	memcpyDoubleArrayToDevice(&W_dev, &W, K + 1);
	t1 = omp_get_wtime();
	for (int i = 0;i < LIMIT; i++)
	{
		
		//do f on all points
		fOnGPUKernel <<<num_blocks, NUM_CUDA_CORES>>> (device_results,points, W_dev, N,K);
		syncAndCheckErrors("fOnGPUKernel launch failed\n");
		
		//find first point to fail for each block
		findFirstIncorrectPointInBlockKernel <<<1, num_blocks >>> (device_results, sum_results,N);
		syncAndCheckErrors("sumResultsKernel launch failed\n");
		
		//adjust W if fault found, output in sum_results[0]
		adjustW_with_faulty_point<<<1,1>>>(sum_results, num_blocks, points, W_dev, W_dev_temp, K, alpha);
		syncAndCheckErrors("adjustW_with_faulty_point launch failed\n");
		
		hipMemcpy(&flag_sum_results, &(sum_results[0]), sizeof(int), hipMemcpyDeviceToHost);
		if (flag_sum_results == ALL_POINTS_CORRECT)
			break;
	}
	
	t2 = omp_get_wtime();
	
	memcpyDoubleArrayToHost(&W, &W_dev, K + 1);
	printf("\nGPU time for alpha %f - %f - W compute\n",alpha,t2-t1);
	/*
	Check quality
	*/
	/********************************************************************************************/
	t1 = omp_get_wtime();
	//Do f on all points with adjusted W
	if (flag_sum_results == W_ADJUSTED)
	{
		fOnGPUKernel << <num_blocks, NUM_CUDA_CORES >> > (device_results, points, W_dev, N, K);
		cudaStatus = hipGetLastError();
		CHECK_ERRORS(cudaStatus, "fOnGPUKernel launch failed\n", hipErrorUnknown)
		cudaStatus = hipDeviceSynchronize();
		CHECK_ERRORS(cudaStatus, "Cuda sync failed\n", hipErrorUnknown)
	}
	/********************************************************************************************
	count number of correct points in each block
	*/
	countCorrectPointsKernel <<<1, num_blocks >>> (device_results, sum_results, N);
	cudaStatus = hipGetLastError();
	CHECK_ERRORS(cudaStatus, "sumResultsKernel launch failed\n", hipErrorUnknown)
	cudaStatus = hipDeviceSynchronize();
	CHECK_ERRORS(cudaStatus, "Cuda sync failed\n", hipErrorUnknown)
	/********************************************************************************************
	count of incorrect points in sum_results[0]
	*/
	sumCountResultsKernel <<<1, 1>> >(sum_results, num_blocks);
	cudaStatus = hipGetLastError();
	CHECK_ERRORS(cudaStatus, "adjustW_with_faulty_point launch failed\n", hipErrorUnknown)
	cudaStatus = hipDeviceSynchronize();
	CHECK_ERRORS(cudaStatus, "Cuda sync failed\n", hipErrorUnknown)
	/********************************************************************************************/
	int count;
	hipMemcpy(&count, &(sum_results[0]), sizeof(int), hipMemcpyDeviceToHost);
	*q = (count / (double) N);
	t2 = omp_get_wtime();
	printf("\nGPU time for alpha %f - %f - q compute\n", alpha, t2 - t1);
	return cudaStatus;
}