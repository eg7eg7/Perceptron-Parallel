#include "hip/hip_runtime.h"

#include <stdio.h>
#include <omp.h>
#include "myMacro.h"
#include "myApp.h"
#include <math.h>
#include "cudaKernel.h"
#include "Perceptron.h"

__device__ double fOnGPU(int i) {

	return 1;
}


__global__ void sumResultsKernel(int *result, int *sum_results, int size) {
	int i, index = threadIdx.x;
	sum_results[index] = POINT_CORRECT;
	int chunk_size = NUM_CUDA_CORES;
	int start_index = threadIdx.x * chunk_size;

	for (i = start_index; i < start_index + chunk_size; i++) {
		if (i >= size)
			break;
		if (result[i] != POINT_CORRECT) {
			sum_results[index] = result[i];
			break;
		}
	}

}

__device__ void mult_scalar_with_vector_device(double* vector, int dim, double scalar, double* result_vector) {
	for (int i = 0; i < dim; i++)
		result_vector[i] = vector[i] * scalar;
}

__device__ void add_vector_to_vector_device(double* vector1, double* vector2, int dim, double* result_vector) {
	for (int i = 0; i < dim; i++)
		result_vector[i] = vector1[i] + vector2[i];
}

__device__ void device_adjustW(double* W, double* temp_vector, Point* point, int K, double alpha) {
	double val = mult_vector_with_vector_device((*point).x, W, K + 1);
	int sign;
	if (val >= 0)
		sign = SET_A;
	else
		sign = SET_B;

	mult_scalar_with_vector_device((*point).x, K + 1, alpha*(-sign), temp_vector);
	add_vector_to_vector_device(W, temp_vector, K + 1, W);

}
//return 1 if all points correct, return -1 if W is adjusted
__global__ void adjustW_with_faulty_point(int *faulty_points,int size,Point* points, double* W,double* temp_vector,int K,double alpha) {
	int index;
	for (int i = 0; i < size; i++)
	{
		index = faulty_points[i];
		if (index != POINT_CORRECT)
		{
			//ADJUST W and return 
			device_adjustW(W,temp_vector, &(points[index]),K,alpha);
			faulty_points[0] = W_ADJUSTED;
			return;
		}
	}
	faulty_points[0] = ALL_POINTS_CORRECT;
}

__device__ double mult_vector_with_vector_device(double* vector1, double* vector2, int dim) {
	double result = vector1[0] * vector2[0];
	for (int i = 1; i < dim; i++)
		result += vector1[i] * vector2[i];
	return result;
}

__global__ void fOnGPUKernel(int *result, Point* points,double* W, int N,int K) {
	int index = threadIdx.x + blockIdx.x * NUM_CUDA_CORES;
	if (index >= N)
		return;
	double val = mult_vector_with_vector_device(points[index].x, W, K+1);
	if (val*points[index].set < 0)
		result[index] = index;
	else
		result[index] = POINT_CORRECT;

}


hipError_t CopyPointsToDevice(Point* points, Point** dev_points,double*** dev_x_points, int N, int K) {
	*dev_x_points = (double**)malloc(sizeof(double*)*N);
	
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipSetDevice(0);
	CHECK_ERRORS(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?", hipErrorUnknown)

		
	cudaStatus = hipMalloc((void**)dev_points, N * sizeof(Point));
	CHECK_ERRORS(cudaStatus, "hipMalloc failed!", hipErrorUnknown)

#pragma omp for
		for (int i = 0; i < N; i++)
		{
			cudaStatus = hipMalloc((void**)&((*dev_x_points)[i]), (K + 1) * sizeof(double));
			CHECK_ERRORS(cudaStatus, "hipMalloc failed!", hipErrorUnknown);
			hipMemcpy((*dev_x_points)[i], points[i].x, sizeof(double)*(K+1), hipMemcpyHostToDevice);
			Point pt;
			pt.x = (*dev_x_points)[i];
			pt.set = points[i].set;
			hipMemcpy(&(*dev_points)[i], &pt, sizeof(Point), hipMemcpyHostToDevice);
		}
	return cudaStatus;
}

hipError_t freePointsFromDevice(Point** dev_points, double*** dev_x_points, int N) {
	hipError_t cudaStatus = hipSuccess;
	cudaStatus = hipSetDevice(0);
	CHECK_ERRORS(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?", hipErrorUnknown)
		// Allocate GPU buffer for temporary results - one member for each thread.
		for (int i = 0; i < N; i++)
		{
			cudaStatus = hipFree((*dev_x_points)[i]);
			CHECK_ERRORS(cudaStatus, "hipFree failed!", hipErrorUnknown)
		}
	cudaStatus = hipFree(*dev_points);
	CHECK_ERRORS(cudaStatus, "hipFree failed!", hipErrorUnknown)
	free(*dev_x_points);
	return cudaStatus;
}

hipError_t get_quality_with_alpha_GPU(Point* points, double alpha, double* W, int N, int K, int LIMIT) {
	int* device_results;
	int* sum_results;
	double t1, t2;
	hipError_t cudaStatus = hipSuccess;
	double *W_dev,*W_dev_temp;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	CHECK_ERRORS(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?", hipErrorUnknown)
	cudaStatus = hipMalloc((void**)&W_dev, sizeof(double)*(K + 1));
	CHECK_ERRORS(cudaStatus, "hipMalloc failed!", hipErrorUnknown)
	hipMemcpy(W_dev, W, sizeof(double)*(K + 1), hipMemcpyHostToDevice);

	cudaStatus = hipMalloc((void**)&W_dev_temp, sizeof(double)*(K + 1));
	CHECK_ERRORS(cudaStatus, "hipMalloc failed!", hipErrorUnknown)

	cudaStatus = hipMalloc((void**)&device_results, sizeof(int)*N);
	CHECK_ERRORS(cudaStatus, "hipMalloc failed!", hipErrorUnknown)
	
	t1 = omp_get_wtime();
	int num_blocks = (int) ceil(N / (double) NUM_CUDA_CORES);
	cudaStatus = hipMalloc((void**)&sum_results, sizeof(int)*num_blocks);
	CHECK_ERRORS(cudaStatus, "hipMalloc failed!", hipErrorUnknown)


	for (int i = 0;i < LIMIT; i++)
	{
		/*
		do f on all points
		*/
	fOnGPUKernel <<<num_blocks, NUM_CUDA_CORES>>> (device_results,points, W_dev, N,K);
	cudaStatus = hipGetLastError();
	CHECK_ERRORS(cudaStatus, "fOnGPUKernel launch failed", hipErrorUnknown)
	cudaStatus = hipDeviceSynchronize();
	CHECK_ERRORS(cudaStatus, "Cuda sync failed", hipErrorUnknown)
	/*
	find first point to fail
	*/
	sumResultsKernel <<<1, num_blocks >>> (device_results, sum_results,N);
	CHECK_ERRORS(cudaStatus, "sumResultsKernel launch failed", hipErrorUnknown)
	cudaStatus = hipDeviceSynchronize();
	CHECK_ERRORS(cudaStatus, "Cuda sync failed", hipErrorUnknown)
	/*
	adjust W if fault found
	*/
	adjustW_with_faulty_point<<<1,1>>>(sum_results, num_blocks, points, W_dev, W_dev_temp, K, alpha);
	cudaStatus = hipDeviceSynchronize();
	CHECK_ERRORS(cudaStatus, "Cuda sync failed", hipErrorUnknown)
		if (sum_results[0] == ALL_POINTS_CORRECT)
			break;
	}
	
	t2 = omp_get_wtime();
	hipMemcpy(W_dev, W, sizeof(double)*(K + 1), hipMemcpyDeviceToHost);
	hipFree(W_dev);
	hipFree(W_dev_temp);
	hipFree(device_results);
	hipFree(sum_results);
	printf("\nGPU time = %f\n", t2 - t1);
	return cudaStatus;
}